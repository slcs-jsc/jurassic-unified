#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <omp.h>
#include "jr_common.h" // ...

#ifdef GPUDEBUG
    #define debug_printf(...) printf(__VA_ARGS__)
#else
    #define debug_printf(...)
#endif

	// Helper /////////////////////////////////////////////////////////////////////
	// Checking return types of all CUDA runtime functions is best practice, 
	//  ... has negligible performance impact and should not be omitted unless absolutely necessary
	__host__ inline
	void __cudaSafeCall(hipError_t err, const char *file, const int line, char const *call=nullptr) { // Actual check function
		if (hipSuccess != err) {
			fprintf(stderr, "[ERROR] CUDA call%s%s at %s:%d\n%s\n", call?" to ":"", call, file, line, hipGetErrorString(err));
			exit(0);
		}
	} // __cudaSafeCall
    #define cuCheck(err) __cudaSafeCall((err), __FILE__, __LINE__, #err) // Syntactic sugar to enhance output

    // As CUDA kernel launches are asynchronous error checking is more difficult, 
    // ... as the check might occur prior to the actual error - this macro makes 
    // ... sure it catches an error if it occurs by explicit Synchronization. 
    // ... Due to the performance impact it is only active in debug mode.
    __host__ inline
    void __cuKernelCheck(const char* file, const int line) {
#ifdef GPUDEBUG
		hipDeviceSynchronize();
		hipError_t err = hipPeekAtLastError();
		if (hipSuccess != err) {
			fprintf(stderr, "[ERROR] CUDA kernel call at %s:%d\n%s\n",  file, line, hipGetErrorString(err));
			exit(0);
		} // err
#endif
	} // __cuKernelCheck
    #define cuKernelCheck() __cuKernelCheck(__FILE__, __LINE__)

	// GPU Memory management /////////////////////////////////////////////////////////

    __host__
    void copy_data_to_GPU(void *d, void const *h, size_t const nBytes, hipStream_t const stream) {
        debug_printf("[INFO] transfer %lu Byte from %p @host to %p @device\n", nBytes, h, d);
        cuCheck(hipMemcpyAsync(d, h, nBytes, hipMemcpyHostToDevice, stream));
    } // copy_data_to_GPU

    __host__
    void get_data_from_GPU(void *h, void const *d, size_t const nBytes, hipStream_t const stream) {
        debug_printf("[INFO] transfer %lu Byte from %p @device to %p @host\n", nBytes, d, h);
        cuCheck(hipMemcpyAsync(h, d, nBytes, hipMemcpyDeviceToHost, stream));
    } // get_data_from_GPU

    __host__
    void* __allocate_on_GPU(size_t const nBytes, char const *srcfile=nullptr, int const srcline=0) {
        debug_printf("[INFO] hipMalloc %.6f MByte in %s:%i\n", 1e-6*nBytes, srcfile, srcline);
        void* d = nullptr;
        cuCheck(hipMalloc(&d, nBytes));
        return d;
    } // allocate_on_GPU
    #define malloc_GPU(TYPE, NUM) (TYPE *)__allocate_on_GPU((NUM)*sizeof(TYPE), __FILE__, __LINE__)

    __host__
    void free_memory_on_GPU(void**d) {
        cuCheck(hipFree(*d));
        *d = nullptr;
    } // free_memory_on_GPU

    
    __host__
    void* __allocate_unified_memory(size_t const nBytes, char const *srcfile=nullptr, int const srcline=0) {
        debug_printf("[INFO] hipMallocManaged %.6f MByte in %s:%i\n", 1e-6*nBytes, srcfile, srcline);
        void* d = nullptr;
        cuCheck(hipMallocManaged(&d, nBytes));
        return d;
    } // allocate_on_GPU
    #define getUnifiedMemory(TYPE, NUM) (TYPE *)__allocate_unified_memory((NUM)*sizeof(TYPE), __FILE__, __LINE__)
    
  extern "C" {
	  trans_table_t* get_tbl_on_GPU(ctl_t const *ctl); 
  }
    __host__
	trans_table_t* get_tbl_on_GPU(ctl_t const *ctl) {
    printf("DEBUG #%d get_tbl_on_GPU was called..\n", ctl->MPIglobrank);
		static trans_table_t *tbl_G = nullptr;
		if (!tbl_G) {
      printf("DEBUG #%d tbl_G == nullptr\n", ctl->MPIglobrank);
			trans_table_t* tbl = get_tbl(ctl);
#ifdef  USE_UNIFIED_MEMORY_FOR_TABLES
            printf("[INFO] allocated %.3f MByte unified memory for tables\n", 1e-6*sizeof(trans_table_t));
            tbl_G = tbl; // just passing a pointer, same memory space
#else
            printf("[INFO] try to allocate %.3f MByte GPU memory for tables\n", 1e-6*sizeof(trans_table_t));
			tbl_G = malloc_GPU(trans_table_t, 1);
			copy_data_to_GPU(tbl_G, tbl, sizeof(trans_table_t), 0);
#endif
		} // !tbl_G
		return tbl_G;
	} // get_tbl_on_GPU

	// ################ GPU driver routines - keep consistent with CPUdrivers.cu ##############

	// Radiance -> Brightness conversion //////////////////////////////////////////
	void __global__ // GPU-kernel
		radiance_to_brightness_GPU(ctl_t const *ctl, obs_t *obs) { // operates onto obs in-place
			for(int ir = blockIdx.x; ir < obs->nr; ir += gridDim.x) { // grid stride loop over blocks = rays
				for(int id = threadIdx.x; id < ctl->nd; id += blockDim.x) { // grid stride loop over threads = detectors
                    auto const radiance = obs->rad[ir][id];
					obs->rad[ir][id] = brightness_core(radiance, ctl->nu[id]); // modify in-place
				} // id
			} // ir
		} // radiance_to_brightness_GPU

	// Add planetary surface emission ////////////////////////////////////////////
	void __global__ // GPU-kernel
		surface_terms_GPU(const trans_table_t *tbl, obs_t *obs, double const tsurf[], int const nd) {
			for(int ir = blockIdx.x; ir < obs->nr; ir += gridDim.x) { // grid stride loop over blocks = rays
				for(int id = threadIdx.x; id < nd; id += blockDim.x) { // grid stride loop over threads = detectors
					add_surface_core(obs, tbl, tsurf[ir], ir, id);
				} // id
			} // ir
		} // surface_terms_GPU

  template<int CO2, int H2O, int N2, int O2>
  __host__ __device__ __ext_inline__
  double continua_core_temp(ctl_t const *ctl, pos_t const *los, int const ig_co2, int const ig_h2o, int const id) {
      double const p = los->p;
      double const t = los->t;
      double const ds = los->ds;
      double beta_ds = los->k[ctl->window[id]]*ds;													// extinction
      // make sure that ig_co2 and ig_h2o are both >= 0
      if ((CO2)) beta_ds += continua_ctmco2(ctl->nu[id], p, t, los->u[ig_co2]);						// co2 continuum
      if ((H2O)) beta_ds += continua_ctmh2o(ctl->nu[id], p, t, los->q[ig_h2o], los->u[ig_h2o]);		// h2o continuum
      if ((N2))  beta_ds += continua_ctmn2(ctl->nu[id], p, t)*ds;									// n2 continuum
      if ((O2))  beta_ds += continua_ctmo2(ctl->nu[id], p, t)*ds;									// o2 continuum
      return     beta_ds;
  } // continua_core_bbbb where each b is either 0 or 1
  
  template<int CO2, int H2O, int N2, int O2>
    void __global__ // GPU-kernel
    fusion_kernel_GPU(trans_table_t const *tbl, ctl_t const *ctl,
        obs_t *obs, pos_t const (*restrict los)[NLOS],
        int const np[], int const ig_co2, int const ig_h2o,
        double const (*restrict aero_beta)[ND]) {
			double tau_path[NG];
			for(int ir = blockIdx.x; ir < obs->nr; ir += gridDim.x) { // grid stride loop over blocks = rays
				for(int id = threadIdx.x; id < ND; id += blockDim.x) { // grid stride loop over threads = detectors
					obs->rad[ir][id] = 0.0;
					obs->tau[ir][id] = 1.0;
				} // id
				for(int ig = 0; ig < NG; ++ig) {
					tau_path[ig] = 1.0;
				} // ig
				for(int ip = 0; ip < np[ir]; ++ip) {
					for(int id = threadIdx.x; id < ctl->nd; id += blockDim.x) { // grid stride loop over threads = detectors
						double const beta_ds = continua_core_temp <CO2, H2O, N2, O2>
                                   (ctl, &(los[ir][ip]), ig_co2, ig_h2o, id);           // function args
						
            //Added:
            double aero_ds = 0;
            if(NULL != aero_beta) // only if scattering is included
              aero_ds = los[ir][ip].aerofac * aero_beta[los[ir][ip].aeroi][id] * los[ir][ip].ds;
            
            double const tau_gas = apply_ega_core(tbl, &(los[ir][ip]), tau_path, ctl->ng, id);
						double const planck = src_planck_core(tbl, los[ir][ip].t, id);
						new_obs_core(obs, ir, id, beta_ds + aero_ds, planck, tau_gas);
					} // id --> parallel over detectors=threads
				} // ip --> non-parallelisable
			} // ir --> parallel over rays==blocks
    } // fusion_kernel_GPU

    __host__
	void multi_version_GPU(char const fourbit, trans_table_t const *tbl, ctl_t const *ctl,
			obs_t *obs, pos_t const (*restrict los)[NLOS],
			int const np[], int const ig_co2, int const ig_h2o,
      double const (*restrict aero_beta)[ND],
			unsigned const grid, unsigned const block, unsigned const shmem, hipStream_t const stream) {
#define LaunchKernel <<< grid, block, shmem, stream >>> (tbl, ctl, obs, los, np, ig_co2, ig_h2o, aero_beta)
		switch (fourbit) {
			case 0b0000: fusion_kernel_GPU<0,0,0,0> LaunchKernel; break;
			case 0b0001: fusion_kernel_GPU<0,0,0,1> LaunchKernel; break;
			case 0b0010: fusion_kernel_GPU<0,0,1,0> LaunchKernel; break;
			case 0b0011: fusion_kernel_GPU<0,0,1,1> LaunchKernel; break;
			case 0b0100: fusion_kernel_GPU<0,1,0,0> LaunchKernel; break;
			case 0b0101: fusion_kernel_GPU<0,1,0,1> LaunchKernel; break;
			case 0b0110: fusion_kernel_GPU<0,1,1,0> LaunchKernel; break;
			case 0b0111: fusion_kernel_GPU<0,1,1,1> LaunchKernel; break;
			case 0b1000: fusion_kernel_GPU<1,0,0,0> LaunchKernel; break;
			case 0b1001: fusion_kernel_GPU<1,0,0,1> LaunchKernel; break;
			case 0b1010: fusion_kernel_GPU<1,0,1,0> LaunchKernel; break;
			case 0b1011: fusion_kernel_GPU<1,0,1,1> LaunchKernel; break;
			case 0b1100: fusion_kernel_GPU<1,1,0,0> LaunchKernel; break;
			case 0b1101: fusion_kernel_GPU<1,1,0,1> LaunchKernel; break;
			case 0b1110: fusion_kernel_GPU<1,1,1,0> LaunchKernel; break;
			case 0b1111: fusion_kernel_GPU<1,1,1,1> LaunchKernel; break;
		} // fourbit
#undef	LaunchKernel
	} // multi_version_GPU

	// Raytracing ////////////////////////////////////////////////////////////////
  template<int scattering_included>
	void __global__ // GPU-kernel
		raytrace_rays_GPU(ctl_t const *ctl, const atm_t *atm, obs_t *obs, pos_t
    los[][NLOS], double *tsurf, int np[], int const *atm_id, aero_t const *aero) {
			for(int ir = blockIdx.x*blockDim.x + threadIdx.x; ir < obs->nr; ir += blockDim.x*gridDim.x) { // grid stride loop over rays
        np[ir] = traceray<scattering_included>(ctl, &atm[(NULL == atm_id ? 0 : atm_id[ir])], obs, ir, los[ir], &tsurf[ir], aero);
			} // ir
		} // raytrace_rays_GPU

	// Compute hydrostatic equilibria for all atm //////////////////////////////
	void __global__ // GPU-kernel
		hydrostatic_kernel_GPU(ctl_t const *ctl, atm_t *atm, const int num_of_atms, int const ig_h2o) {
			for(int i = blockIdx.x*blockDim.x + threadIdx.x; i < num_of_atms; i += blockDim.x*gridDim.x) {
				hydrostatic_1d_h2o(ctl, &atm[i], 0, atm[i].np, ig_h2o);
			} // ip
		} // hydrostatic_kernel

    __host__
	void hydrostatic1d_GPU(ctl_t const *ctl, ctl_t const *ctl_G,
			atm_t *atm_G, int const num_of_atms, int const ig_h2o, hipStream_t const stream) {
		if(ctl->hydz < 0) return; // Check reference height
		hydrostatic_kernel_GPU<<<num_of_atms/32 + 1, 32, 0, stream>>> (ctl_G, atm_G, num_of_atms, ig_h2o);
	} // hydrostatic1d_GPU

	// ################ end of GPU driver routines ##############

	// GPU control struct containing GPU version of input, intermediate and output arrays
	typedef struct {
    aero_t *aero_G; // new
		obs_t  *obs_G;
		atm_t  *atm_G;
		pos_t (*los_G)[NLOS];
		double *tsurf_G;
		int    *np_G;
    double (*aero_beta_G)[ND];
    int *atm_id_G; 
		hipStream_t stream;
	} gpuLane_t;

	// The full forward model working on one package of NR rays
	void formod_one_package_GPU(ctl_t *ctl, ctl_t *ctl_G,
			trans_table_t const *tbl_G,
			atm_t *atm, // can be made const if we do not get the atms back
			obs_t *obs,
      aero_t const *aero,
      int const *atm_id,
			gpuLane_t const *gpu)
    // a workload manager for the GPU
    {
		debug_printf("[INFO] %s GPU\n"
               " Rays:    %9d (max %d)\n"
               " Gases:   %9d (max %d)\n"
               " Channels:%9d (max %d)\n",
               __func__, obs->nr, NR, ctl->ng, NG, ctl->nd, ND);
        
		atm_t *atm_G = gpu->atm_G;
    int *atm_id_G = gpu->atm_id_G;
    aero_t *aero_G = gpu->aero_G;
		obs_t *obs_G = gpu->obs_G;
    double (*aero_beta_G)[ND] = gpu->aero_beta_G;
    pos_t (* los_G)[NLOS] = gpu->los_G;
		double *tsurf_G = gpu->tsurf_G;
		int *np_G = gpu->np_G;
	  hipSetDevice(0);
    hipEvent_t finishedEvent;
		hipEventCreate(&finishedEvent);

		// gas absorption continua configuration
		static int ig_co2 = -999, ig_h2o = -999;
		if((ctl->ctm_h2o) && (ig_h2o == -999)) ig_h2o = jur_find_emitter(ctl, "H2O");
		if((ctl->ctm_co2) && (ig_co2 == -999)) ig_co2 = jur_find_emitter(ctl, "CO2");
		// binary switches for the four gases
		char const fourbit = (char)
                ( ( (1 == ctl->ctm_co2) && (ig_co2 >= 0) )*0b1000   // CO2
                + ( (1 == ctl->ctm_h2o) && (ig_h2o >= 0) )*0b0100   // H2O
                +   (1 == ctl->ctm_n2)                    *0b0010   // N2
                +   (1 == ctl->ctm_o2)                    *0b0001); // O2

		unsigned const nd = ctl->nd, nr = obs->nr; // abbreviate
    int const num_of_atms = jur_get_num_of_atms(nr, atm_id);

    // "beta_a" -> 'a', "beta_e" -> 'e'
    char const beta_type = ctl->sca_ext[5];
		
    hipStream_t stream = gpu->stream;

		copy_data_to_GPU(obs_G, obs, 1*sizeof(obs_t), stream);
    if(NULL != aero) { // only if scattering is included
      copy_data_to_GPU(aero_G, aero, 1*sizeof(aero_t), stream);
      copy_data_to_GPU(aero_beta_G, ('a' == beta_type) ? aero->beta_a :
                       aero->beta_e, NLMAX * ND * sizeof(double), stream);
    }
    if(NULL != atm_id) {
      copy_data_to_GPU(atm_id_G, atm_id, nr * sizeof(int), stream);
		  copy_data_to_GPU(atm_G, atm, num_of_atms * sizeof(atm_t), stream);
    }
    else {
      atm_id_G = NULL;
		  copy_data_to_GPU(atm_G, atm, sizeof(atm_t), stream);
    }
        
		hydrostatic1d_GPU(ctl, ctl_G, atm_G, num_of_atms, ig_h2o, stream); // in this call atm_G gets modified
		cuKernelCheck();

    if(NULL != aero && ctl->sca_n > 0) { // only if scattering is included
      raytrace_rays_GPU<1> <<< (nr/64)+1, 64, 0, stream>>> (ctl_G, atm_G, obs_G, los_G, tsurf_G, np_G, atm_id_G, aero_G);
      cuKernelCheck();
    } else {
      raytrace_rays_GPU<0> <<< (nr/64)+1, 64, 0, stream>>> (ctl_G, atm_G, obs_G, los_G, tsurf_G, np_G, atm_id_G, NULL);
      cuKernelCheck();
    }
	
    multi_version_GPU(fourbit, tbl_G, ctl_G, obs_G, los_G, np_G, ig_co2, ig_h2o,
                      NULL == aero ? NULL:  aero_beta_G,
                      nr, nd, ctl->gpu_nbytes_shared_memory, stream);
		cuKernelCheck();

    surface_terms_GPU <<< nr, nd, 0, stream>>> (tbl_G, obs_G, tsurf_G, nd);
		cuKernelCheck();
        
    if (ctl->write_bbt && ctl->leaf_nr == -1) { // convert radiance to brightness (in-place)
        radiance_to_brightness_GPU <<< nr, nd, 0, stream >>> (ctl_G, obs_G);
    } // write_bbt

// 		get_data_from_GPU(atm, atm_G, 1*sizeof(atm_t), stream); // do we really need to get the atms back?
		get_data_from_GPU(obs, obs_G, 1*sizeof(obs_t), stream); // always transfer NR rays
    
    printf("Heisenbug :) %lf\n", obs->rad[0][0]);

		// Wait for GPU operations to complete
		cuCheck(hipEventRecord(finishedEvent, stream));
		cuCheck(hipEventSynchronize(finishedEvent));
	} // formod_one_package_GPU

    // make sure that jur_formod_multiple_packages_GPU can be linked from CPUdrivers.c
	extern "C" {
	   void jur_formod_multiple_packages_GPU(ctl_t *ctl, atm_t *atm, obs_t *obs,
                                           int n, int const *atm_id, aero_t const *aero);
   }

	__host__
	void jur_formod_multiple_packages_GPU(ctl_t *ctl, atm_t *atm, obs_t *obs,
                                        int n, int const *atm_id, aero_t const *aero) {
    static ctl_t *ctl_G=NULL;
		static trans_table_t *tbl_G=NULL;

		static int numDevices = 0;
		static gpuLane_t* gpuLanes=NULL;
		static size_t numLanes = 0;
		// static size_t nextLane = 0;
		// size_t myLane = 0;

		static bool do_init = true;

    // it can also by set to true, the initial value is irelevant
    static bool multi_atm_before = false;
    const bool multi_atm_now = NULL != atm_id;

    atm_t **divided_atms;
    int **divided_atm_ids;

    if(multi_atm_now) {
      divided_atms = (atm_t **) malloc(n * sizeof(atm_t *));
      divided_atm_ids = (int **) malloc(n * sizeof(int *));
      jur_divide_atm_data_into_packages(atm, obs, n, atm_id, divided_atms, divided_atm_ids);
    }

#pragma omp critical
		{
			if (do_init || (!do_init && multi_atm_before != multi_atm_now)) {
              double tic = omp_get_wtime();
              size_t sizePerLane = sizeof(aero_t) + sizeof(obs_t) + sizeof(atm_t) + NR * (NLOS * sizeof(pos_t) + sizeof(double) + sizeof(int));
              // in this case we have NR * atm_t instead of the only one and one additional atm_id array
              if(multi_atm_now)
                sizePerLane = sizeof(aero_t) + sizeof(obs_t) + NR * (sizeof(atm_t) + NLOS * sizeof(pos_t) + sizeof(double) + 2 * sizeof(int));
 
              if (ctl->checkmode) {
                printf("# %s: GPU memory requirement per lane is %.3f MByte\n", __func__, 1e-6*sizePerLane);
              } else {
                hipSetDevice(0);
                cuCheck(hipGetDeviceCount(&numDevices));

                if(!do_init && multi_atm_before != multi_atm_now) {
                  for(size_t lane = 0; lane < numLanes; ++lane) {
                    gpuLane_t* gpu = &(gpuLanes[lane]); // abbreviation
                    // Deallocation of GPU memory
                    free_memory_on_GPU((void **) &gpu->aero_G);
                    free_memory_on_GPU((void **) &gpu->obs_G);
                    free_memory_on_GPU((void **) &gpu->atm_G);
                    free_memory_on_GPU((void **) &gpu->tsurf_G);
                    free_memory_on_GPU((void **) &gpu->np_G);
                    free_memory_on_GPU((void **) &gpu->los_G);
                    free_memory_on_GPU((void **) &gpu->aero_beta_G);
                    if(multi_atm_before)
                      free_memory_on_GPU((void **) &gpu->atm_id_G);
                  }
                  free(gpuLanes);
                  debug_printf("[INFO] multi_atm_now = %d\n", (int) multi_atm_now);
                  debug_printf("[INFO] GPU lanes memory deallocation\n");
                }

                if(do_init) {
                  // Initialize ctl and tbl-struct (1 per GPU)
                  ctl_G = malloc_GPU(ctl_t, 1);
                  copy_data_to_GPU(ctl_G, ctl, sizeof(ctl_t), 0);
                  double tic = omp_get_wtime();
                  tbl_G = get_tbl_on_GPU(ctl);
                  double toc = omp_get_wtime();
                  printf("TIMER #%d jurassic-gpu reading table time: %lf\n",
                  ctl->MPIglobrank, toc - tic);
                }

                // Get number of possible lanes
                size_t gpuMemFree, gpuMemTotal;
                cuCheck(hipMemGetInfo(&gpuMemFree, &gpuMemTotal));
                debug_printf("[INFO] memory GPU: free %.3f of total %.3f MByte = %.1f %%\n",
                              1e-6*gpuMemFree, 1e-6*gpuMemTotal, gpuMemFree/(.01*gpuMemTotal));

                numLanes = (size_t)((0.9*gpuMemFree) / (double)sizePerLane); // Only use 90% of free GPU memory ...
                printf("DEBUG #%d max possible number of Lanes: %d\n",
                ctl->MPIglobrank, numLanes);
                // ... other space is needed for alignment and profiling buffers
                size_t const maxNumLanes = 4; // Do not really need more than a handfull of lanes
                if (numLanes > maxNumLanes) numLanes = maxNumLanes;
                debug_printf("[INFO] GPU memory per lane: %.3f MByte, try to fit %i lanes\n", 1e-6*sizePerLane, numLanes);
                if (numLanes < 1) ERRMSG("Memory requirement per lane is too high, no lanes");

                gpuLanes = (gpuLane_t*) malloc(numLanes*sizeof(gpuLane_t)); // (this memory is never freed)
                for(size_t lane = 0; lane < numLanes; ++lane) {
                  gpuLane_t* gpu = &(gpuLanes[lane]); // abbreviation
                  // Allocation of GPU memory
                  gpu->aero_G		   = malloc_GPU(aero_t, 1);
                  gpu->obs_G		   = malloc_GPU(obs_t, 1);
                  gpu->tsurf_G	   = malloc_GPU(double, NR);
                  gpu->np_G		     = malloc_GPU(int, NR);
                  gpu->los_G		   = (pos_t (*)[NLOS])__allocate_on_GPU(NR*NLOS*sizeof(pos_t), __FILE__, __LINE__); 
                  gpu->aero_beta_G = (double(*)[ND])__allocate_on_GPU(NLMAX*ND*sizeof(double), __FILE__, __LINE__);
                  if(multi_atm_now) {
                    gpu->atm_id_G  = malloc_GPU(int, NR);
                    gpu->atm_G     = malloc_GPU(atm_t, NR);
                  }
                  else {
                    gpu->atm_id_G  = NULL;
                    gpu->atm_G	   = malloc_GPU(atm_t, 1);
                  }
                  cuCheck(hipStreamCreate(&gpu->stream));
                  debug_printf("[INFO] hipStreamCreate --> streamId %d\n", gpu->stream);
                } // lane
                debug_printf("[INFO] multi_atm_now = %d\n", (int) multi_atm_now);
                debug_printf("[INFO] GPU lanes memory allocation\n");
                cuCheck(hipMemGetInfo(&gpuMemFree, &gpuMemTotal));
                debug_printf("[INFO] memory GPU: free %.3f of total %.3f MByte = %.1f %%\n",
                              1e-6*gpuMemFree, 1e-6*gpuMemTotal, gpuMemFree/(.01*gpuMemTotal));
              } // checkmode

				do_init = false;
        multi_atm_before = multi_atm_now;
        double toc = omp_get_wtime();
        printf("TIMER #%d jurassic-gpu gpu_lanes initialization time: %lf\n", ctl->MPIglobrank, toc - tic);
			} // do_init || (!do_init && multi_atm_before != multi_atm_now)
    } //<------- omp critical is here, maybe I should put it to the end of the function!

    // TODO: this may be a probelm in juwels-booster case (with more then one GPU  device per node)
    hipSetDevice(0);

    if (ctl->checkmode) { printf("# %s: no operation in checkmode\n", __func__); return; }
    
    printf("numDevices: %d\n", numDevices);
    printf("DEBUG #%d numDevices: %d\n", ctl->MPIglobrank, numDevices);

    // I had add to this because of CPUs converting los to pos..    
    omp_set_nested(true);
#pragma omp parallel num_threads(numLanes)
#pragma omp for schedule(dynamic, 1) //work stealing
    for(int i = 0; i < n; i++) //loop over packages
    {
      int const myLane = omp_get_thread_num();
      assert(myLane < numLanes);
      char mask[NR][ND];
      save_mask(mask, &obs[i], ctl);
      copy_data_to_GPU(ctl_G, ctl, sizeof(ctl_t), gpuLanes[myLane].stream); // controls might change, update
      if(multi_atm_now)
        formod_one_package_GPU(ctl, ctl_G, tbl_G, divided_atms[i], &obs[i], aero, divided_atm_ids[i], &gpuLanes[myLane]);
      else
        formod_one_package_GPU(ctl, ctl_G, tbl_G, atm, &obs[i], aero, atm_id, &gpuLanes[myLane]);
      apply_mask(mask, &obs[i], ctl);
    }
    omp_set_nested(false);
    if(multi_atm_now) {
      for(int i = 0; i < n; i++) {
        free(divided_atms[i]);
        free(divided_atm_ids[i]);
      }
      free(divided_atms);
      free(divided_atm_ids);
    }
	} // jur_formod_multiple_packages_GPU
